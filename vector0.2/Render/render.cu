#include "hip/hip_runtime.h"


#ifndef _RENDER_H_
#define _RENDER_H_

#include <stdio.h>
#include <stdlib.h>
#include "radial.h"

using Radial::Vec2d;

#define IMAGE_WIDTH 512
#define IMAGE_HEIGHT 512
#define PI 3.1416f
#define INF 9999
#define ZERO 0.000001

/* #ifndef UNIT */
/* #define UNIT */
typedef unsigned int unit;
/* #endif */

/* Position enum */
enum Position {LEFT, RIGHT, MIDDLE};

__device__ float2 operator+(float2 l, float2 r){
     float2 res;
     res.x = l.x+r.x;
     res.y = l.y + r.y;
     return res;
}

__device__ float2 operator-(float2 l, float2 r){
     float2 res;
     res.x = l.x-r.x;
     res.y = l.y - r.y;
     return res;
}

__device__ float operator*(float2 l, float2 r){
     return l.x*r.x + l.y*r.y;
}

__device__ float length(float2 l){
     return sqrt(l.x*l.x + l.y*l.y);
}

__device__ bool operator<(float3 &res, float r){
     return res.x < r && res.y < r && res.z < r;
}

__device__ bool operator>(float3 &res, float r){
     return res.x > r && res.y > r && res.z > r;
}

/* "operator=" must be a member function */

/* __device__ float3 operator=(float3 &res, float r){ */
/*      res.x = r; */
/*      res.y = r; */
/*      res.z = r; */
/*      return res; */
/* } */

__device__ float3 setValue(float3 &res, float r){
     res.x = r;
     res.y = r;
     res.z = r;
     return res;
}

__device__ float3 operator+=(float3& res, float3 r){
     res.x += r.x;
     res.y += r.y;
     res.z += r.z;
     return res;
}

__device__ float3 operator/=(float3& res, float r){
     res.x /= r;
     res.y /= r;
     res.z /= r;
     return res;
}
__device__ float3 operator-(float3& p1, float3& p2){
     float3 res;
     res.x = p1.x - p2.x;
     res.y = p1.y - p2.y;
     res.z = p1.z - p2.z;
     return res;
}
__device__ float3 operator+(float3 p1, float3 p2){
     float3 res;
     res.x = p1.x+p2.x;
     res.y = p1.y+p2.y;
     res.z = p1.z+p2.z;
     return res;
}
__device__ float operator*(float3 p1, float3 p2){
     return p1.x*p2.x + p1.y*p2.y + p1.z*p2.z;
}

__device__ float3 operator*(float3 p1, float k){
     float3 res = p1;
     res.x *= k;
     res.y *= k;
     res.z *= k;
     return res;
}

__device__ float3 operator/(float3 p1, float k){
     float3 res = p1;
     res.x /= k;
     res.y /= k;
     res.z /= k;
     return res;
}
__device__ float dot(float3 p1, float3 p2){
     return p1.x*p2.x + p1.y*p2.y + p1.z*p2.z;
}

__device__ float cross(float3 p1, float3 p2){
     return p1.x*p2.y - p1.y*p2.x;
}

__device__ float length(float3 p){
     return sqrt(p.x*p.x + p.y*p.y + p.z*p.z);
}

__device__ float3 norm(float3& p){
     float l = length(p);
     p.x /= l;
     p.y /= l;
     p.z /= l;
     return p;
}


__device__ float3 perpendicular(float3 p){
     float3 res;
     res.x = p.y;
     res.y = -p.x;
     res.z = p.z;
     return res;
}

__device__ float length_square(float3 p){
     return p.x*p.x + p.y*p.y + p.z*p.z;
}

__device__ float distance_p2p(float3 p1, float3 p2){
     float3 p = p1-p2;
     return length(p);
}

__device__ float distance_p2l(float3 p, float3 lstart, float3 lend){
     float3 op = p - lstart;
     float3 ol = lend - lstart;
     float f = op * ol / length_square(ol);
     if(f < 0) return distance_p2p(p, lstart);
     if(f > 1) return distance_p2p(p, lend);
     return perpendicular(ol) * op / length(ol);
}

__device__ bool isIntersect(float3 o, float3 dir, float3 lstart, float3 lend){
     float3 PaQa = make_float3(lstart.x - o.x, lstart.y - o.y, 0);
     float3 PaQb = make_float3(lend.x - o.x, lend.y - o.y, 0);
     float3 PaPb = make_float3(dir.x * 1000, dir.y * 1000, 0);

     float res = cross(PaQa, PaPb) * cross(PaPb, PaQb);
     if( res >= ZERO ){
          return true;
     }else
          return false;
}

__device__ float distance_v2l(float3 o, float3 dir, float3 lstart, float3 lend){
     if(isIntersect(o, dir, lstart, lend) == false)
          return INF;
     // Store the values for fast access and easy
     float3 p1(o), p2(o+dir*1000), p3(lstart), p4(lend);
// equations-to-code conversion
     float x1 = p1.x, x2 = p2.x, x3 = p3.x, x4 = p4.x;
     float y1 = p1.y, y2 = p2.y, y3 = p3.y, y4 = p4.y;
 
     float d = (x1 - x2) * (y3 - y4) - (y1 - y2) * (x3 - x4);
// If d is zero, there is no intersection
     if (d == 0) return INF;
 
// Get the x and y
     float pre = (x1*y2 - y1*x2), post = (x3*y4 - y3*x4);
     float x = ( pre * (x3 - x4) - (x1 - x2) * post ) / d;
     float y = ( pre * (y3 - y4) - (y1 - y2) * post ) / d;
 
// Check if the x and y coordinates are within both lines
     if ( x < min(x1, x2) || x > max(x1, x2) ||
          x < min(x3, x4) || x > max(x3, x4) ) return INF;
     if ( y < min(y1, y2) || y > max(y1, y2) ||
          y < min(y3, y4) || y > max(y3, y4) ) return INF;
 
// Return the point of intersection
     return sqrt( (o.x-x)*(o.x-x) + (o.y-y)*(o.y-y) );
}

__device__ Position position_p2l(float3 p, float3 lstart, float3 lend){
     float3 op = p - lstart;
     float3 ol = lend - lstart;
     if(perpendicular(ol) * op < 0)
          return LEFT;
     else
          return RIGHT;
}

__device__ float3 coordinate(float3 p, float3 lstart, float3 lend){
     float3 op = p - lstart;
     float3 ol = lend - lstart;
     // u为y v为x 平方进行归一化
     float u = op * ol / length_square(ol);
     float v = op * perpendicular(ol) / length_square(ol);
     float3 res = make_float3(v, u, 0.f);
     return res;
}

__device__ float gauss(float x, float a=0.5f){
     return 1.f/sqrt(2*PI)*exp(-x*x/a);
}


/* Kernel */
__global__ void add(int width,  int height, float *d){
     int x = blockIdx.x*blockDim.x + threadIdx.x;
     int y = blockIdx.y*blockDim.y + threadIdx.y;
     if(x<width && y<height){
          d[y*height+x] = 1.0;
     }

}

__global__ void test(){
     float3 test0 = make_float3(0.f, 0.f, 0.f);
     float3 test1 = make_float3(0.1f, 0.1f, 0.1f);
     
}


__device__ void
diffusePix(int x, int y, float3 *imageSource, float3 *imageRes, bool *constrained){
     /* constrained */
     if(constrained[y*IMAGE_WIDTH + x] == true){
          imageRes[y*IMAGE_WIDTH + x].x = imageSource[y*IMAGE_WIDTH + x].x;
          imageRes[y*IMAGE_WIDTH + x].y = imageSource[y*IMAGE_WIDTH + x].y;
          imageRes[y*IMAGE_WIDTH + x].z = imageSource[y*IMAGE_WIDTH + x].z;
     }
     
     /* diffuse if not constrained */
     int pixcount = 0;
     imageRes[y*IMAGE_WIDTH + x] = make_float3(0.f, 0.f, 0.f);
     
     /* left: x-1*/
     if(x > 0){
          imageRes[y*IMAGE_WIDTH + x] += imageSource[y*IMAGE_WIDTH + x-1];
          pixcount ++;
     }
     /* right: x+1 */
     if(x < IMAGE_WIDTH-1){
          imageRes[y*IMAGE_WIDTH + x] += imageSource[y*IMAGE_WIDTH + x+1];
          pixcount++;
     }
     /* up: y-1 */
     if(y > 0){
          imageRes[y*IMAGE_WIDTH + x] += imageSource[(y-1)*IMAGE_WIDTH + x];
          pixcount++;
     }
     /* down: y+1 */
     if(y < IMAGE_HEIGHT-1){
          imageRes[y*IMAGE_WIDTH + x] += imageSource[(y+1)*IMAGE_WIDTH + x];
          pixcount++;
     }

     /* calculate the averaged values and store  */
     imageRes[y*IMAGE_WIDTH + x] /= pixcount;
          
}

__global__ void
diffuse(float3 *imageSource, float3 *imageRes, bool *constrained){
     int x = blockIdx.x*blockDim.x + threadIdx.x;
     int y = blockIdx.y*blockDim.y + threadIdx.y;
     if(x<IMAGE_WIDTH && y<IMAGE_HEIGHT){
          diffusePix(x, y, imageSource, imageRes, constrained);
     }
}

__global__ void
updateConstrain(bool *constrained, float3 *imageSource){
     int x = blockIdx.x*blockDim.x + threadIdx.x;
     int y = blockIdx.y*blockDim.y + threadIdx.y;
     if(imageSource[y*IMAGE_WIDTH + x] < 0.00001) /* is 0.0f */
          constrained[y*IMAGE_WIDTH + x] = false;
     else
          constrained[y*IMAGE_WIDTH + x] = true;
}
__device__ float weightPix(float3 p, float3 lstart, float3 lend){
     float res;
     res = gauss( distance_p2l(p, lstart, lend)/105 );
     return res;
}

__device__ void
distancePix(int x, int y, int lineSize, float3 *distMap,
         float3 *pointStart, float3 *pointEnd){

     float3 pix = make_float3(x, y, 0.f);
     float3 color = make_float3(1.f, 1.f, 1.f);
     float dis = INF;
     int id = 0;
     for(int i = 0; i < lineSize; ++i)
     {
          if( dis > distance_p2l(pix, pointStart[i], pointEnd[i]) ){
               dis = distance_p2l(pix, pointStart[i], pointEnd[i]);
               id = i;
          }
     }
     
     distMap[(IMAGE_HEIGHT-1-y)*IMAGE_WIDTH +x] = color * (20.f * id);

}

__global__ void
distanceMap(int lineSize, float3 *distMap, float3 *pointStart, float3 *pointEnd)
{
     int x = blockIdx.x*blockDim.x + threadIdx.x;
     int y = blockIdx.y*blockDim.y + threadIdx.y; 
     if(x<IMAGE_WIDTH && y<IMAGE_HEIGHT){
          distancePix(x, y, lineSize, distMap,
                   pointStart, pointEnd);
     }
}     
__device__ void
fieldPix(int x, int y, int lineSize,
         float3 *imageRes,
         float3 *pointStart, float3 *pointEnd, float3 *pointVector,
         float3 *leftColor, float3 *rightColor){
     /* for each pix */
     float3 pix = make_float3(x, y, 0.f);
     float3 colorSum = make_float3(.0f, .0f, .0f);
     float weightSum = .0f;
     for(int i = 0; i < lineSize; ++i)
     {
          /* weight */
          float w = weightPix(pix, pointStart[i], pointEnd[i]);
          weightSum += w;
          /* set color */
          if(position_p2l(pix, pointStart[i], pointEnd[i]) == LEFT)
               colorSum += leftColor[i] * w;
          else
               colorSum += rightColor[i] * w;
     }

     //colorSum /= weightSum;
     if(colorSum > 255.f)
          setValue(colorSum, 255.f);
     imageRes[(IMAGE_HEIGHT-1-y)*IMAGE_WIDTH +x] = colorSum; /* 坐标上下置换 */
}


__global__ void
field(float3 *imageRes, int lineSize,
      float3 *pointStart, float3 *pointEnd, float3 *pointVector,
      float3 *leftColor, float3 *rightColor)
{
     int x = blockIdx.x*blockDim.x + threadIdx.x;
     int y = blockIdx.y*blockDim.y + threadIdx.y; 
     if(x<IMAGE_WIDTH && y<IMAGE_HEIGHT){
          fieldPix(x, y, lineSize, imageRes,
                   pointStart, pointEnd, pointVector,
                   leftColor, rightColor);
     }
}
__device__ float weight_v2l(float3 o, float3 dir, float3 lstart, float3 lend){
     float p = -0.;
     float dis = distance_v2l(o, dir, lstart, lend);
     if(dis >= INF-1) return 0;
     return pow(dis, p);
}
__device__ int relation_p2l(float3 o, float3 dir, float3 lstart, float3 lend){
     return 0;
}
__device__ void
rayTracingPix(int x, int y, int lineSize, int _rayNum,
         float3 *imageRes,
         float3 *pointStart, float3 *pointEnd, float3 *pointVector,
         float3 *leftColor, float3 *rightColor)
{
     /* for each pix */
     int rayNum = _rayNum;
     if(rayNum < 0 || rayNum > 10000)
          printf("========== rayNum reset : %d -> 360 ==========\n"), rayNum = 360;
     
     float3 colorSum = make_float3(0, 0, 0);
     for(int k = 0; k < rayNum; ++k)
     {
           // each degree
          float3 o = make_float3(x, y, 0);
          float3 dir = make_float3(cos(k/2/PI), sin(k/2/PI), 0);
          int line = -1;
          int left_or_right = -1;
          float w = 0;
          float dis = INF-1;

          for(int l = 0; l < lineSize; ++l)
          {
               float temp = distance_v2l(o, dir, pointStart[l], pointEnd[l]);
               if(temp < dis){
                    dis = temp;
                    left_or_right = position_p2l(o, pointStart[l], pointEnd[l]);
                    w = weight_v2l(o, dir, pointStart[l], pointEnd[l]);
                    line = l;
               }
               if(line >= 0 && l < lineSize){
                    if(left_or_right == RIGHT){
                         colorSum += rightColor[line] * w;
                    }
                    else if(left_or_right == LEFT){
                         colorSum += leftColor[line] * w;
                    }
                    else{
                         ;      /* in the line do nothing */
                    }
               }
          }
     }
     imageRes[(IMAGE_HEIGHT-1-y)*IMAGE_WIDTH +x] = colorSum / rayNum;
     
}
__global__ void
rayTracing(float3 *imageRes, int lineSize, int rayNum,
      float3 *pointStart, float3 *pointEnd, float3 *pointVector,
      float3 *leftColor, float3 *rightColor)
{
     int x = blockIdx.x*blockDim.x + threadIdx.x;
     int y = blockIdx.y*blockDim.y + threadIdx.y; 
     if(x<IMAGE_WIDTH && y<IMAGE_HEIGHT){
          rayTracingPix(x, y, lineSize, rayNum, imageRes,
                   pointStart, pointEnd, pointVector,
                   leftColor, rightColor);
     }
}     

/* extern */

extern "C" void
cudaWriteImage(char* filename, int imageSize, float3 *imageRes)
{
     FILE *fp = fopen(filename, "w");
     fprintf(fp, "P3\n%d %d\n%d\n", IMAGE_WIDTH, IMAGE_HEIGHT, 255);
     for(int i=0; i<imageSize; ++i){
          fprintf(fp,  "%d %d %d ", (int)imageRes[i].x, (int)imageRes[i].y, (int)imageRes[i].z);
     }
     fclose(fp);
}

extern "C" void
cudaDiffuseRender(int **image)
{
     printf("Cuda Diffuse Start.....\n");
     float3 *imageSource, *imageRes, *imageTmp;
     bool *constrained;
     unsigned int imageSize = IMAGE_WIDTH * IMAGE_HEIGHT;
     hipMalloc((void**)&imageSource,
                imageSize*sizeof(float3));
     hipMalloc((void**)&imageRes,
                imageSize*sizeof(float3));
     hipMalloc((void**)&constrained,
                imageSize*sizeof(bool));
     imageTmp = new float3[imageSize];

     /* printf("Malloc Succeed\n"); */
     
     /* initialize */
     for(int i=0; i<imageSize; ++i){
          imageTmp[i].x = image[i][0];
          imageTmp[i].y = image[i][1];
          imageTmp[i].z = image[i][2];
     }
     hipMemcpy(imageSource, imageTmp, imageSize*sizeof(float3), hipMemcpyHostToDevice);
     
     /* printf("Initialize Succeed\n"); */
     /* set block */
     dim3 block (16,16,1);
     dim3 grid(IMAGE_WIDTH/16,IMAGE_HEIGHT/16,1);

     int it = 100;
     for(int i=0; i<it; ++i){
          updateConstrain<<<grid,block>>>(constrained, imageSource);
          diffuse<<<grid,block>>>(imageSource, imageRes, constrained);
          hipMemcpy(imageSource, imageRes, imageSize*sizeof(float3), hipMemcpyDeviceToDevice); /* frome res -> source */
          //printf("\r%d iterator Succeed...", i+1);
     }
     /* test<<<grid,block>>>(); */

     /* write to file */
     printf("Write to file...\n");
     hipMemcpy(imageTmp, imageRes, imageSize*sizeof(float3), hipMemcpyDeviceToHost);
     FILE *fp = fopen("image_diffuse.ppm", "w");
     fprintf(fp, "P3\n%d %d\n%d\n", IMAGE_WIDTH, IMAGE_HEIGHT, 255);
     for(int i=0; i<imageSize; ++i){
          fprintf(fp,  "%d %d %d ", (int)imageTmp[i].x, (int)imageTmp[i].y, (int)imageTmp[i].z);
     }
     fclose(fp);
     //printf("Succeed...\n");

     hipFree(imageSource);
     hipFree(imageRes);
     hipFree(constrained);
     delete[] imageTmp;
     return;
}


extern "C" void
cudaFieldRender(float **ps, float **pe, float **pv, int **lc, int **rc, int **iimage, int size){
     /* Malloc */
     float3 *pointStart, *pointEnd, *pointVector;
     float3 *leftColor, *rightColor;
     float3 *imageRes;
     float3 *imageDist;
     int lineSize = size, imageSize = IMAGE_WIDTH*IMAGE_HEIGHT;
     hipMalloc((void**)&pointStart,lineSize*sizeof(float3));
     hipMalloc((void**)&pointEnd,lineSize*sizeof(float3));
     hipMalloc((void**)&pointVector,lineSize*sizeof(float3));
     hipMalloc((void**)&leftColor,lineSize*sizeof(float3));
     hipMalloc((void**)&rightColor,lineSize*sizeof(float3));
     hipMalloc((void**)&imageRes,imageSize*sizeof(float3));
     hipMalloc((void**)&imageDist,imageSize*sizeof(float3));
     
     /* Initialize */
     float3 *tmp = new float3[lineSize];
     float3 *imageTmp = new float3[imageSize];
     /* pointStart */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = ps[i][0];
          tmp[i].y = ps[i][1];
          tmp[i].z = ps[i][2];
     }
     hipMemcpy(pointStart, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);
     /* pointEnd */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = pe[i][0];
          tmp[i].y = pe[i][1];
          tmp[i].z = pe[i][2];
     }
     hipMemcpy(pointEnd, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);
     /* pointVector */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = pv[i][0];
          tmp[i].y = pv[i][1];
          tmp[i].z = pv[i][2];
     }
     hipMemcpy(pointVector, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);
     /* leftColor */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = lc[i][0];
          tmp[i].y = lc[i][1];
          tmp[i].z = lc[i][2];
     }
     hipMemcpy(leftColor, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);
     /* rightColor */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = rc[i][0];
          tmp[i].y = rc[i][1];
          tmp[i].z = rc[i][2];
     }
     hipMemcpy(rightColor, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);

     /* Initialize Over */
     printf("Total %d Line \n", lineSize);
     /* Render */
     
     /* set block and do */
     dim3 block (16,16,1);
     dim3 grid(IMAGE_WIDTH/16,IMAGE_HEIGHT/16,1);

     distanceMap<<<grid, block>>>(lineSize, imageDist, pointStart, pointEnd);
     field<<<grid, block>>>(imageRes, lineSize, pointStart, pointEnd, pointVector, leftColor, rightColor);

     
     /* feed back result image */
     hipMemcpy(imageTmp, imageDist, imageSize*sizeof(float3), hipMemcpyDeviceToHost);
     cudaWriteImage("imageDist.ppm",imageSize, imageTmp);
     
     hipMemcpy(imageTmp, imageRes, imageSize*sizeof(float3), hipMemcpyDeviceToHost);

     for(int i = 0; i < imageSize; ++i)
     {
          iimage[i][0] = imageTmp[i].x;
          iimage[i][1] = imageTmp[i].y;
          iimage[i][2] = imageTmp[i].z;
     }

     cudaDiffuseRender(iimage);
     /* free */
     hipFree(pointStart);
     hipFree(pointEnd);
     hipFree(pointVector);
     hipFree(leftColor);
     hipFree(rightColor);
     hipFree(imageRes);
     hipFree(imageDist);
     delete[] tmp;
     delete[] imageTmp;
}

extern "C" void
cudaRayTracingRender(float **ps, float **pe, float **pv, int **lc, int **rc, int rayNum, int **iimage, int size) /* iimage return to user */
{
     /* Malloc */
     float3 *pointStart, *pointEnd, *pointVector;
     float3 *leftColor, *rightColor;
     float3 *imageRes;
     int lineSize = size, imageSize = IMAGE_WIDTH*IMAGE_HEIGHT;
     hipMalloc((void**)&pointStart,lineSize*sizeof(float3));
     hipMalloc((void**)&pointEnd,lineSize*sizeof(float3));
     hipMalloc((void**)&pointVector,lineSize*sizeof(float3));
     hipMalloc((void**)&leftColor,lineSize*sizeof(float3));
     hipMalloc((void**)&rightColor,lineSize*sizeof(float3));
     hipMalloc((void**)&imageRes,imageSize*sizeof(float3));
     
     /* Initialize */
     float3 *tmp = new float3[lineSize];
     float3 *imageTmp = new float3[imageSize];
     /* pointStart */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = ps[i][0];
          tmp[i].y = ps[i][1];
          tmp[i].z = ps[i][2];
     }
     hipMemcpy(pointStart, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);
     /* pointEnd */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = pe[i][0];
          tmp[i].y = pe[i][1];
          tmp[i].z = pe[i][2];
     }
     hipMemcpy(pointEnd, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);
     /* pointVector */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = pv[i][0];
          tmp[i].y = pv[i][1];
          tmp[i].z = pv[i][2];
     }
     hipMemcpy(pointVector, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);
     /* leftColor */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = lc[i][0];
          tmp[i].y = lc[i][1];
          tmp[i].z = lc[i][2];
     }
     hipMemcpy(leftColor, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);
     /* rightColor */
     for(int i = 0; i < lineSize; ++i)
     {
          tmp[i].x = rc[i][0];
          tmp[i].y = rc[i][1];
          tmp[i].z = rc[i][2];
     }
     hipMemcpy(rightColor, tmp, lineSize*sizeof(float3), hipMemcpyHostToDevice);

     /* Initialize Over */
     printf("Total %d Line \n", lineSize);


     
     /* Render */
     
     /* set block and do */
     dim3 block (16,16,1);
     dim3 grid(IMAGE_WIDTH/16,IMAGE_HEIGHT/16,1);

     rayTracing<<<grid, block>>>(imageRes, lineSize, rayNum, pointStart, pointEnd, pointVector, leftColor, rightColor);

     
     /* feed back result image */
     hipMemcpy(imageTmp, imageRes, imageSize*sizeof(float3), hipMemcpyDeviceToHost);

     for(int i = 0; i < imageSize; ++i)
     {
          iimage[i][0] = imageTmp[i].x;
          iimage[i][1] = imageTmp[i].y;
          iimage[i][2] = imageTmp[i].z;
     }

     /* free */
     hipFree(pointStart);
     hipFree(pointEnd);
     hipFree(pointVector);
     hipFree(leftColor);
     hipFree(rightColor);
     hipFree(imageRes);
     delete[] tmp;
     delete[] imageTmp;
}     

#endif /* _RENDER_H_ */
